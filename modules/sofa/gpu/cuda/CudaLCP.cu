#include "hip/hip_runtime.h"
#include "CudaCommon.h"
#include "CudaMath.h"
#include <stdio.h>

#if defined(__cplusplus)
namespace sofa
{
namespace gpu
{
namespace cuda
{
#endif

extern "C"
{
	void CudaLCP_MultVector(int dim,int index, const void * m,const void * f,void * r);
	int CudaLCP_MultVector_ResSize(unsigned int dim);
	void CudaLCP_ComputeError(int compteur2,int sizeTmp, const void * tmp, const void * M,const void * q,void * f,void * res,void * error);
}

__global__ void CudaLCP_MultVector_kernel(int dim, int i, const float * m,const float * f, float * r, int offset) {
	//! Dynamically allocated shared memory for gather
	extern  __shared__  float temp[];
	int index0 = umul24(blockIdx.x,blockDim.x);
	int index1 = threadIdx.x;
	int n = blockDim.x; //min(blockDim.x , size-index0);
	float acc = 0;
	int index = index0+index1;
	if (index < dim && index != i)
		acc = m[index] * f[index];

	while(offset>0)
	{
		if (index1 >= offset && index1 < n)
			temp[index1] = acc;
		__syncthreads();
		if (index1+offset < n)
			acc += temp[index1+offset];
		n = offset;
		offset >>= 1;
	}
	if (index1 == 0)
		r[blockIdx.x] = acc;
}

__global__ void CudaLCP_ComputeError_kernel(int compteur2, const float * tmp,const float * M, const float * q,float * f, float * res, float * error, int offset) {
	//! Dynamically allocated shared memory for gather
	extern  __shared__  float temp[];
	int index1 = threadIdx.x;
	int n = blockDim.x;
	float r;
	
	r = tmp[index1];
	
	while(offset>0)
	{
		if (index1 >= offset && index1 < n)
			temp[index1] = r;
		__syncthreads();
		if (index1+offset < n)
			r += temp[index1+offset];
		n = offset;
		offset >>= 1;
	}
	
	if (index1==0) {
		r += q[compteur2];
		//for (int i=0;i<sizeTmp;++i)	r += tmp[i];			
		res[compteur2] = r;
		
		float f_1 = f[compteur2];

		float f_new;
		float MindM = M[0];
		
		if (r<0) f_new=-r/MindM;
		else f_new=0.0;
			
		f[compteur2] = f_new;
		
		error[0] += fabs(MindM * (f_new - f_1) );		
	}
}

int CudaLCP_MultVector_ResSize(unsigned int dim) {
	return (dim+BSIZE-1)/BSIZE;
}

void CudaLCP_MultVector(int dim,int index, const void * m,const void * f,void * r) {
	dim3 threads(BSIZE,1);
	dim3 grid((dim+BSIZE-1)/BSIZE,1);

	CudaLCP_MultVector_kernel<<< grid, threads, threads.x*sizeof(float) >>>(dim, index, (const float*)m, (const float*)f, (float*)r, BSIZE/2);
}

void CudaLCP_ComputeError(int compteur2,int sizeTmp, const void * tmp, const void * M,const void * q,void * f,void * res,void * error) {
	dim3 threads(sizeTmp,1);
	dim3 grid(1,1);
	int offset;
	if (sizeTmp==1)
		offset = 0;
	else
	{
		offset = 1;
		while (offset*2 < sizeTmp)
			offset *= 2;
	}

	CudaLCP_ComputeError_kernel<<< grid, threads, threads.x*sizeof(float) >>>(compteur2,(const float*)tmp, (const float*)M,(const float*)q, (float*)f, (float*)res,(float*)error,offset);
}

#if defined(__cplusplus)
} // namespace cuda
} // namespace gpu
} // namespace sofa
#endif
