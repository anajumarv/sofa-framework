#include "hip/hip_runtime.h"
/******************************************************************************
*       SOFA, Simulation Open-Framework Architecture, version 1.0 beta 3      *
*                (c) 2006-2008 MGH, INRIA, USTL, UJF, CNRS                    *
*                                                                             *
* This library is free software; you can redistribute it and/or modify it     *
* under the terms of the GNU Lesser General Public License as published by    *
* the Free Software Foundation; either version 2.1 of the License, or (at     *
* your option) any later version.                                             *
*                                                                             *
* This library is distributed in the hope that it will be useful, but WITHOUT *
* ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or       *
* FITNESS FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public License *
* for more details.                                                           *
*                                                                             *
* You should have received a copy of the GNU Lesser General Public License    *
* along with this library; if not, write to the Free Software Foundation,     *
* Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301 USA.          *
*******************************************************************************
*                               SOFA :: Modules                               *
*                                                                             *
* Authors: The SOFA Team and external contributors (see Authors.txt)          *
*                                                                             *
* Contact information: contact@sofa-framework.org                             *
******************************************************************************/
#include "CudaCommon.h"
#include "CudaMath.h"
#include "hip/hip_runtime.h"

#if defined(__cplusplus) && CUDA_VERSION != 2000
namespace sofa
{
namespace gpu
{
namespace cuda
{
#endif

extern "C"
{
void CudaTestForceField3f_addForce(unsigned int nbElem, unsigned int nbVertex, unsigned int nbElemPerVertex, const void* elems, void* state, const void* velems, void* f, const void* x, const void* v);
void CudaTestForceField3f_addDForce(unsigned int nbElem, unsigned int nbVertex, unsigned int nbElemPerVertex, const void* elems, void* state, const void* velems, void* df, const void* dx);
}

class __align__(16) GPUElement
{
public:
    /// @name index of the 8 connected vertices
    /// @{
    int v[8];
    /// @}
    /// @name material stiffness
    /// @{
    float Kvol, Kr;
    /// @}
    /// @name initial volume and radius
    /// @{
    float initvol, initr;
    /// @}
};

class __align__(16) GPUElementState
{
public:
    /// center
    CudaVec3<float> center;
    /// volume diff*Kvol
    float dv;
    float r,Kr;
    float dummy1,dummy2;
};

//////////////////////
// GPU-side methods //
//////////////////////

#define getX(i) (((const CudaVec3<float>*)x)[i])

__global__ void CudaTestForceField3f_calcForce_kernel(int nbElem, const GPUElement* elems, GPUElementState* state, const float* x)
{
    int index0 = umul24(blockIdx.x,BSIZE); //blockDim.x;
    int index1 = threadIdx.x;
    int index = index0+index1;

    GPUElement e = elems[index];

    GPUElementState s;

    if (index < nbElem)
    {
        s.center = getX(e.v[0]);
	s.center += getX(e.v[1]);
	s.center += getX(e.v[2]);
	s.center += getX(e.v[3]);
	s.center += getX(e.v[4]);
	s.center += getX(e.v[5]);
	s.center += getX(e.v[6]);
	s.center += getX(e.v[7]);
	s.center *= 0.125f;
	s.dv = 0;
	s.r = e.initr;
	s.Kr = e.Kr;
    }

    state[index] = s;

}

__global__ void CudaTestForceField3f_addForce_kernel(int nbVertex, unsigned int nbElemPerVertex, const GPUElement* elems, GPUElementState* state, const int* velems, float* f, const float* x)
{
    int index0 = umul24(blockIdx.x,BSIZE); //blockDim.x;
    int index1 = threadIdx.x;
    int index3 = umul24(index1,3); //3*index1;

        //! Dynamically allocated shared memory to reorder global memory access
    extern  __shared__  float temp[];

    // First copy x inside temp
    int iext = umul24(blockIdx.x,BSIZE*3)+index1; //index0*3+index1;

    temp[index1        ] = x[iext        ];
    temp[index1+  BSIZE] = x[iext+  BSIZE];
    temp[index1+2*BSIZE] = x[iext+2*BSIZE];

    __syncthreads();

    CudaVec3<float> pos1 = CudaVec3<float>::make(temp[index3  ],temp[index3+1],temp[index3+2]);

    CudaVec3<float> force = CudaVec3<float>::make(0.0f,0.0f,0.0f);

    velems+=index0*nbElemPerVertex+index1;

    if (index0+index1 < nbVertex)
    for (int s = 0;s < nbElemPerVertex; s++)
    {
        int i = *velems -1;
        velems+=BSIZE;
	if (i != -1)
	{
            int eindex = i >> 3; // element index
            i &= 7;              // vertice index inside the element
            //GPUElement e = elems[eindex];
            GPUElementState s = state[eindex];
	    CudaVec3<float> dp = pos1 - s.center;
	    float inv_r = invnorm(dp);
	    float r = __fdividef(1.0,inv_r);
	    float dr = (s.r-r)*s.Kr*inv_r;
	    force += dp*dr;

	}
    }

    __syncthreads();

    temp[index3  ] = force.x;
    temp[index3+1] = force.y;
    temp[index3+2] = force.z;

    __syncthreads();

    f[iext        ] += temp[index1        ];
    f[iext+  BSIZE] += temp[index1+  BSIZE];
    f[iext+2*BSIZE] += temp[index1+2*BSIZE];
}

__global__ void CudaTestForceField3f_calcDForce_kernel(int nbElem, const GPUElement* elems, GPUElementState* state, const float* x)
{
    int index0 = umul24(blockIdx.x,BSIZE); //blockDim.x;
    int index1 = threadIdx.x;
    int index = index0+index1;

    //GPUElement e = elems[index];
    GPUElementState s = state[index];

    if (index < nbElem)
    {
    }

    state[index] = s;

}

//////////////////////
// CPU-side methods //
//////////////////////

void CudaTestForceField3f_addForce(unsigned int nbElem, unsigned int nbVertex, unsigned int nbElemPerVertex, const void* elems, void* state, const void* velems, void* f, const void* x, const void* v)
{
    dim3 threads1(BSIZE,1);
    dim3 grid1((nbElem+BSIZE-1)/BSIZE,1);
    CudaTestForceField3f_calcForce_kernel<<< grid1, threads1>>>(nbElem, (const GPUElement*)elems, (GPUElementState*)state, (const float*)x);
    dim3 threads2(BSIZE,1);
    dim3 grid2((nbVertex+BSIZE-1)/BSIZE,1);
    CudaTestForceField3f_addForce_kernel<<< grid2, threads2, BSIZE*3*sizeof(float) >>>(nbVertex, nbElemPerVertex, (const GPUElement*)elems, (GPUElementState*)state, (const int*)velems, (float*)f, (const float*)x);
}

void CudaTestForceField3f_addDForce(unsigned int nbElem, unsigned int nbVertex, unsigned int nbElemPerVertex, const void* elems, void* state, const void* velems, void* df, const void* dx)
{
    dim3 threads1(BSIZE,1);
    dim3 grid1((nbElem+BSIZE-1)/BSIZE,1);
    //CudaTestForceField3f_calcDForce_kernel<<< grid1, threads1>>>(nbElem, (const GPUElement*)elems, (GPUElementState*)state, (const float*)dx);
    dim3 threads2(BSIZE,1);
    dim3 grid2((nbVertex+BSIZE-1)/BSIZE,1);
    //CudaTestForceField3f_addForce_kernel<<< grid2, threads2, BSIZE*3*sizeof(float) >>>(nbVertex, nbElemPerVertex, (const GPUElement*)elems, (GPUElementState*)state, (const int*)velems, (float*)df, (const float*)dx);
}

#if defined(__cplusplus) && CUDA_VERSION != 2000
} // namespace cuda
} // namespace gpu
} // namespace sofa
#endif
