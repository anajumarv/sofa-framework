#include "mycuda.h"

#if defined(__cplusplus)
namespace sofa
{
namespace gpu
{
namespace cuda
{
#endif

void cudaCheck(hipError_t err, const char* src="?")
{
    if (err == hipSuccess) return;
    //fprintf(stderr, "CUDA: Error %d returned from %s.\n",(int)err,src);
    mycudaLogError(err, src);
}

int mycudaInit(int device)
{
    int deviceCount = 0;
    cudaCheck(hipGetDeviceCount(&deviceCount));
    myprintf("CUDA: %d devices found.\n", deviceCount);
    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t dev;
        dev.name=NULL;
        dev.bytes=0;
        dev.major=0;
        dev.minor=0;
        cudaCheck(hipGetDeviceProperties(&dev,i));
        myprintf("CUDA:  %d : \"%s\", %d MB, revision %d.%d\n",i,(dev.name==NULL?"":dev.name), dev.bytes/(1024*1024), dev.major, dev.minor);
    }
    if (device >= deviceCount)
    {
        myprintf("CUDA: Device %d not found.\n", device);
        return 0;
    }
    else
    {
        cudaCheck(hipSetDevice(device));
        return 1;
    }
}

void mycudaMalloc(void **devPtr, size_t size)
{
    myprintf("CUDA: malloc(%d).\n",size);
    cudaCheck(hipMalloc(devPtr, size),"hipMalloc");
}

void mycudaFree(void *devPtr)
{
    myprintf("CUDA: free().\n");
    cudaCheck(hipFree(devPtr),"hipFree");
}

void mycudaMemcpyHostToDevice(void *dst, const void *src, size_t count)
{
    cudaCheck(hipMemcpy(dst, src, count, hipMemcpyHostToDevice),"hipMemcpyHostToDevice");
}

void mycudaMemcpyDeviceToDevice(void *dst, const void *src, size_t count)
{
    cudaCheck(hipMemcpy(dst, src, count, hipMemcpyDeviceToDevice),"hipMemcpyDeviceToDevice");
}

void mycudaMemcpyDeviceToHost(void *dst, const void *src, size_t count)
{
    cudaCheck(hipMemcpy(dst, src, count, hipMemcpyDeviceToHost),"hipMemcpyDeviceToHost");
}

#if defined(__cplusplus)
} // namespace cuda
} // namespace gpu
} // namespace sofa
#endif
