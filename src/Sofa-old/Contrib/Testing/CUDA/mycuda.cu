#include "mycuda.h"

// \todo Why mycuda.h is not read correctly...
extern "C" {
    extern int mycudaInit(int device);
    extern void mycudaMalloc(void **devPtr, size_t size);
    extern void mycudaFree(void *devPtr);
//extern void mycudaMemcpy(void *dst, const void *src, size_t count, enum hipMemcpyKind kind);
    extern void mycudaMemcpyHostToDevice(void *dst, const void *src, size_t count);
    extern void mycudaMemcpyDeviceToDevice(void *dst, const void *src, size_t count);
    extern void mycudaMemcpyDeviceToHost(void *dst, const void *src, size_t count);

    extern void mycudaLogError(int err, const char* src);
    extern int myprintf(const char* fmt, ...);
}

void cudaCheck(hipError_t err, const char* src="?")
{
    if (err == hipSuccess) return;
    //fprintf(stderr, "CUDA: Error %d returned from %s.\n",(int)err,src);
    mycudaLogError(err, src);
}

int mycudaInit(int device)
{
    int deviceCount = 0;
    cudaCheck(hipGetDeviceCount(&deviceCount));
    myprintf("CUDA: %d devices found.\n", deviceCount);
    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t dev;
        dev.name=NULL;
        dev.mbytes=0;
        dev.major=0;
        dev.minor=0;
        cudaCheck(hipGetDeviceProperties(&dev,i));
        myprintf("CUDA:  %d : \"%s\", %d MB, revision %d.%d\n",i,(dev.name==NULL?"":dev.name), dev.mbytes, dev.major, dev.minor);
    }
    if (device >= deviceCount)
    {
        myprintf("CUDA: Device %d not found.\n", device);
        return 0;
    }
    else
    {
        cudaCheck(hipSetDevice(device));
        return 1;
    }
}

void mycudaMalloc(void **devPtr, size_t size)
{
    myprintf("CUDA: malloc(%d).\n",size);
    cudaCheck(hipMalloc(devPtr, size),"hipMalloc");
}

void mycudaFree(void *devPtr)
{
    myprintf("CUDA: free().\n");
    cudaCheck(hipFree(devPtr),"hipFree");
}

void mycudaMemcpyHostToDevice(void *dst, const void *src, size_t count)
{
    cudaCheck(hipMemcpy(dst, src, count, hipMemcpyHostToDevice),"hipMemcpyHostToDevice");
}

void mycudaMemcpyDeviceToDevice(void *dst, const void *src, size_t count)
{
    cudaCheck(hipMemcpy(dst, src, count, hipMemcpyDeviceToDevice),"hipMemcpyDeviceToDevice");
}

void mycudaMemcpyDeviceToHost(void *dst, const void *src, size_t count)
{
    cudaCheck(hipMemcpy(dst, src, count, hipMemcpyDeviceToHost),"hipMemcpyDeviceToHost");
}
